
#include <hip/hip_runtime.h>
// Verify that we can parse a simple CUDA file with or without -save-temps
// http://llvm.org/PR22936
// RUN: %clang -nocudainc -nocudalib -Werror -fsyntax-only -c %s
//
// Verify that we pass -x cuda-cpp-output to compiler after 
// preprocessing a CUDA file
// RUN: %clang  -Werror -### -save-temps -c %s 2>&1 | FileCheck %s
// CHECK: "-cc1"
// CHECK: "-E"
// CHECK: "-x" "cuda"
// CHECK-NEXT: "-cc1"
// CHECK: "-x" "cuda-cpp-output"
//
// Verify that compiler accepts CUDA syntax with "-x cuda-cpp-output".
// RUN: %clang -Werror -fsyntax-only -x cuda-cpp-output -c %s

int hipConfigureCall(int, int);
__attribute__((global)) void kernel() {}

void func() {
     kernel<<<1,1>>>();
}

